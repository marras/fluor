#include "hip/hip_runtime.h"
/* CUDA-based free diffusion simulator
   To be used with "fluor" program
   (c) Marek Waligórski 2009-2011       */

//Include some standard headers (mixed C++ and CUDA code)
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
//The CUDA functions are called from a class method
#include "fluorescence.h"

#define PI 3.1415926535897932384626

/* Random number generator
   source: http://http.developer.nvidia.com/GPUGems3/gpugems3_ch37.html
   
   For molecule movement, we need Gaussian-distributed numbers,
   so we take a Combined Tausworthe generator (linear),
   plus a Box-Muller transform into Gaussian distribution.
   
   NOTE: NOT PERFECT! */

/* Tausworthe generator step: S1, S2, S3, and M are all constants, 
   z is part of the private per-thread generator state (~seed).  */
__device__ unsigned TausStep(unsigned &z, int S1, int S2, int S3, unsigned M) {  
    unsigned b=(((z << S1) ^ z) >> S2);
    return z = (((z & M) << S3) ^ b);  
}  

/* Linear Congruential Generator step */
__device__ unsigned LCGStep(unsigned &z, unsigned A, unsigned C)  {  
    return z=(A*z+C);
}

/* Linear generator consists of XORing 3 Tausworthe-generated numbers with one LCG.*/
__device__ float HybridTaus(unsigned *z) {  
   // Combined period is lcm(p1,p2,p3,p4)~ 2^121  
    return 2.3283064365387e-10 * (              // Periods  
     TausStep(z[0], 13, 19, 12, 4294967294UL) ^  // p1=2^31-1  
     TausStep(z[1], 2, 25, 4, 4294967288UL) ^    // p2=2^30-1  
     TausStep(z[2], 3, 11, 17, 4294967280UL) ^   // p3=2^28-1  
     LCGStep(z[3], 1664525, 1013904223UL)        // p4=2^32  
    );  
}  


/* Box-Muller transform: get 2 normally-distributed variables from 2 linear vars. */
__device__ float2 BoxMuller(unsigned *z) {  
   float u0=HybridTaus (z), u1=HybridTaus (z);  
   float r=sqrt(-2*log(u0));	//radius
   float theta=2*PI*u1;  	//angle
   return make_float2(r*sin(theta),r*cos(theta));  //here we get the actual (x,y) coordinates
}  

/* Simulate diffusion of molecules on GPU:
	da - positions of molecules
	ds - randomizer seed
	natoms - how many molecules to movement
	cudaDIFF_STEP - avg. movement distance
	size - simulation box size
*/
__global__ void DoTheMovement (float4 *da, unsigned *ds, int natoms, float cudaDIFF_STEP, float3 size) {
    unsigned z[4]={0,0,0,0}; //What the hell? Gdyby zadeklarowac ta zmienna "na zewnatrz" funkcji, to pomimo __device__ i tak bylaby tez widoczna z poziomu hosta!!!

    int moj_nr = blockIdx.x * blockDim.x + threadIdx.x; 	//this number is different for every thread
    z[0]=ds[moj_nr*4];z[1]=ds[1+moj_nr*4];z[2]=ds[2+moj_nr*4];z[3]=ds[3+moj_nr*4]; //faster than a 'for' loop
    
    float2 para = BoxMuller(z);  //get a pair of normally-distributed values

    //move molecules, enable periodic boundary conditions
    da[moj_nr].x += cudaDIFF_STEP * para.x; if (da[moj_nr].x > size.x) da[moj_nr].x -= size.x; else if (da[moj_nr].x < 0) da[moj_nr].x += size.x;
    da[moj_nr].y += cudaDIFF_STEP * para.y; if (da[moj_nr].y > size.y) da[moj_nr].y -= size.y; else if (da[moj_nr].y < 0) da[moj_nr].y += size.y;
    
    para = BoxMuller(z);  //generate next pair of numbers (one of them will get wated :-( )
    da[moj_nr].z += cudaDIFF_STEP * para.x; if (da[moj_nr].z > size.z) da[moj_nr].z -= size.z; else if (da[moj_nr].z < 0) da[moj_nr].z += size.z;
 
    ds[moj_nr*4]=z[0];ds[1+moj_nr*4]=z[1];ds[2+moj_nr*4]=z[2];ds[3+moj_nr*4]=z[3];	 //we have to save the random generator state
}

/* Check excitation of molecules:
	da - positions of molecules
	ds - randomizer seed
	natoms - how many molecules to movement
	dstates - array containing states of each molecule
	epsRAZYdT - probability of excitation at center of confocal volume during this timestep
	F1 - focal point
	SQR_KAPPA - structure parameter of confocal volume (squared)
	WXY - confocal volume radius (XY plane)
*/
__global__ void ExciteMeBaby (float4 *da, unsigned *ds, int natoms, enumStates *dstates, float epsRAZYdT, float3 F1, float SQR_KAPPA, float WXY) { //NOTE we'll try to use the da array already loaded with molecule positions and ds with seeds :)
    int moj_nr = blockIdx.x * blockDim.x + threadIdx.x;
    float4 pos = da[moj_nr];
    
    unsigned z[4]={0,0,0,0}; 
    z[0]=ds[moj_nr*4];z[1]=ds[1+moj_nr*4];z[2]=ds[2+moj_nr*4];z[3]=ds[3+moj_nr*4];
    
    float prob_ex = epsRAZYdT * exp(-2*((pos.x-F1.x)*(pos.x-F1.x)	//probability of excitation (Gaussian profile) [Winkler]
				+(pos.y-F1.y)*(pos.y-F1.y)
				+(pos.z-F1.z)*(pos.z-F1.z)/SQR_KAPPA ) / (WXY*WXY));
    float r1 = HybridTaus(z); 	//here we need a random (0,1)
    if (r1 < prob_ex && dstates[moj_nr] == MS_GROUND) dstates[moj_nr] = MS_EXC_1; //NOTE only excite if molecule is in ground state!

    ds[moj_nr*4]=z[0];ds[1+moj_nr*4]=z[1];ds[2+moj_nr*4]=z[2];ds[3+moj_nr*4]=z[3];	 //save seeds. Optimize somehow?
}


/************************************************************************/
/* CUDA Randomise                                                       */
/************************************************************************/

unsigned *ds; //zarodek randomizera (na karcie graficznej)
float4 *pos, *dpos; //wyniki - pozycje cząstek zmienne z rozkl.norm
enumStates *dstates; //stany czasteczek - na karcie

void Fluorescence :: MoveMolecules_GPU () {
    if (types != 1) {LOG ("!GPU diffusion is not implemented for more than one kind of dye."); return;}
    const float cudaDIFF_STEP = mol[0].DIFFUSION_STEP (); //NOTE THIS ASSUMES ALL MOLECULES HAVE THE SAME SPEED! TODO
    const float3 cudaBox_size = make_float3 (SIZE[0],SIZE[1],SIZE[2]);
    
    DoTheMovement<<<NBlocks,NThreads>>> (dpos, ds, natoms, cudaDIFF_STEP, cudaBox_size); //call the CUDA diffusion kernel
    CUDA_SAFE_CALL( hipDeviceSynchronize() ); 		//wait for GPU to finish calculations
    CUT_CHECK_ERROR("Kernel execution failed\n");
    CUDA_SAFE_CALL( hipMemcpy(pos, dpos, sizeof(float4) * natoms, hipMemcpyDeviceToHost));  //copy data from GPU to "pos" array on CPU

    for (int m=0; m<natoms; m++) {
	mol[m].x[0] = pos[m].x; //update info in Molecule class (TODO optimize)
	mol[m].x[1] = pos[m].y;
	mol[m].x[2] = pos[m].z;
    }
}

void Fluorescence:: TryExcitationALL_GPU () {
  assert (profil == PR_GAUSS && num_foci == 1); // The GPU version is only implemented for 1-focus setup with Gauss excitation.

  const float epsRAZYdT = EPSILON * dT;
  const float3 dF1 = make_float3 (F1[0],F1[1],F1[2]);
  
  CUDA_SAFE_CALL( hipMemcpy(dstates, state, sizeof(enumStates) * natoms, hipMemcpyHostToDevice));  
  
  ExciteMeBaby<<<NBlocks,NThreads>>> (dpos, ds, natoms, dstates, epsRAZYdT, dF1, SQR_KAPPA, WXY); //check which molecules get excited
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  CUT_CHECK_ERROR("Kernel execution failed\n");
  
  CUDA_SAFE_CALL( hipMemcpy(state, dstates, sizeof(enumStates) * natoms, hipMemcpyDeviceToHost)); //read back mol. states from GPU
}

void seed_rng (unsigned *s, int howmany) {	//s- pointer to seed array
    FILE* urandom = fopen( "/dev/urandom", "rb" );
    if (urandom) {
	    register int i = howmany;
	    register bool success = true;
	    while( success && i-- )
		    success = fread( s++, sizeof(unsigned), 1, urandom );
	    fclose(urandom);
    }
    else printf ("ERROR: Failed to open /dev/urandom!\n"); 
}

#ifdef ENABLE_GPU

/* Initialize CUDA */
void Fluorescence :: InitCUDA () {
    int count = 0;
    int i = 0;

    hipGetDeviceCount(&count);		//how many devices are available?
    if(count == 0) {
	    fprintf(stderr, "There is no device.\n");
	    LOG ("!Error initializing CUDA.");
    }

    for(i = 0; i < count; i++) {
	    hipDeviceProp_t prop;
	    if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
		    if(prop.major >= 1) break;
	    }
    }
    if(i == count) {
	    fprintf(stderr, "There is no device supporting CUDA.\n");
	    LOG ("!Error initializing CUDA.");
    }
    hipSetDevice(i);

    printf("CUDA initialized.\n");

    assert (NBlocks*NThreads == natoms); //1 molecule per thread
    
    seed = new unsigned [4*NBlocks*NThreads];
    seed_rng (seed, 4*NBlocks*NThreads); // Generate Threads*4 random numbers (externally, e.g. /dev/urandom) 

    pos = new float4 [natoms]; //host pointer to array of molecule positions
    CUDA_SAFE_CALL (hipMalloc ((void**)&dpos, natoms*sizeof(float4))); //allocate mem for molecule positions (device)
    CUDA_SAFE_CALL (hipMalloc ((void**)&ds, 4*NBlocks*NThreads*sizeof(unsigned))); //and for seeds array (device)

    CUDA_SAFE_CALL (hipMalloc ((void**)&dstates, natoms*sizeof(enumStates))); //zarezerwuj miejsce na stany atomow (device)
    
    for (int m=0; m<natoms; m++) {
	pos[m] = make_float4 (mol[m].x[0],mol[m].x[1],mol[m].x[2],0); //wrap mol. positions into a convenient structure
    } //(float4 is supposed to work faster than float3)
    
    CUDA_SAFE_CALL( hipMemcpy(ds, seed, sizeof(unsigned) * 4*NBlocks*NThreads, hipMemcpyHostToDevice)); //copy seeds to GPU
    CUDA_SAFE_CALL( hipMemcpy(dpos, pos, sizeof(float4) * natoms, hipMemcpyHostToDevice)); //copy starting pos. of molecules
   
    LOG ("---- CUDA ----\nBlocks: %d, Threads: %d.",NBlocks,NThreads);
}

/* Clean up */
void Fluorescence :: DeInitCUDA () {
     CUDA_SAFE_CALL( hipFree(dpos));   
     CUDA_SAFE_CALL( hipFree(ds));   
     delete [] pos;
     delete [] seed;
}


#else //!ENABLE_GPU

void Fluorescence :: InitCUDA () {
	LOG ("*Using CPU only, no CUDA init required.");
}

void Fluorescence :: DeInitCUDA () {
	LOG ("*Using CPU only, no CUDA deinitialization required.");
}
#endif //ENABLE_GPU
